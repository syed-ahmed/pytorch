#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>

__device__ void random_float(float* x) {
    hiprandStatePhilox4_32_10_t state;
    hiprand_init(
            123,
            0,
            4,
            &state);
    auto ret = hiprand_uniform4(&state);
    x[0] = ret.x;
    x[1] = ret.y;
    x[2] = ret.z;
    x[3] = ret.w;
}


__global__ void myKernel(float* x) {
    random_float(x);
}

TEST(DistributionsTest, TestPhiloxIncrement) {
  // Test Description:
  //   In Distributions.cu we mentioned that philox increment
  //   should be at least the number of hiprand() random numbers used in
  //   each thread. In this test, we make sure that uniform_ correctly
  //   increments philox and doesn't reuse randoms from previous calls.
  //    - We check that by first getting 4 randoms from uniform_.
  //      Once we get these 4 randoms, that would mean that philox counter for
  //      thread 0, was incremented by 4.
  //    - Now get 4 randoms with offset=4 from myKernel above.
  //    - Now get 4 more randoms from uniform_ (note thread 0 for this call would
  //      start from a philox_offset value of 4)
  //    - the 4 randoms from myKernel and the 4 randoms from the previous call
  //      of uniform_ should match, signifying that the philox offset was 
  //      incremented properly and no randoms are being reused from previous calls

  // if cuda not available, return
  if (!at::cuda::is_available()) return;

  // manual seed to 123
  at::manual_seed(123);

  // get 4 randoms from uniform_()
  auto self = at::empty({4}, at::TensorOptions(at::kCUDA));
  self.uniform_();

  // allocate 4 float on host memory
  float *x;
  hipMallocManaged(&x, 4*sizeof(float));

  myKernel<<<1, 1>>>(x);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  
  // get 4 new float from uniform_()
  self.uniform_();
  
  // check randoms from myKernel are equal to the randoms from the second
  // call of uniform_()
  for (int i = 0; i < 4; i++) {
    ASSERT_EQ(self[i].item().to<float>(), x[i]);
  }

  // Free memory
  hipFree(x);
}
